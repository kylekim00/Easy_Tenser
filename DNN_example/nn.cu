#include "hip/hip_runtime.h"

#include<stdio.h>
#include<stdlib.h>
#include<string.h>
#include "./../easy_tensor.h"

void print_progress(int count, int max) {
    const int bar_width = 50;

    float progress = (float) count / max;
    int bar_length = progress * bar_width;

    printf("\rProgress: [");
    for (int i = 0; i < bar_length; ++i) {
        printf("#");
    }
    for (int i = bar_length; i < bar_width; ++i) {
        printf(" ");
    }
    printf("] %d / %d", count, max);

    fflush(stdout);
}
int accuracy_CPU(Tensor* O, Tensor* Y){
    if(!O || !Y){
        printf("no Tensor\n");
        return -1;
    }
    if(O->device_type||Y->device_type){
        printf("CPU ONLY\n");
        return -1;
    }
    if(O->num_dim !=2 || Y->num_dim != 1){
        printf("not an appropriate shape.\n");
        return -1;
    }
    if(O->dim[0] != Y->dim[0]){ //batch 비교
        printf("batch size does not match.\n");
        return -1;
    }
    int acc = 0;
    for(int i=0; i < O->dim[0]; i++){
        int max_inx = 0;
        for(int j=1; j < O->dim[1]; j++){
            if (O->T[i * O->stride[0] + j] > O->T[i * O->stride[0] + max_inx]){
                max_inx = j;
            }

        }
        if(max_inx == Y->T[i]){
            acc++;
        }
    }
    return acc;
}
Tensor* copyTensorfromFILE(Tensor* dst, const char* file_name){
    char f_name[50] = "./weight/";
    int len = strlen(f_name);
    int i;
    for(i=0; file_name[i]; i++){
        f_name[i+len] = file_name[i];
    }
    f_name[i+len] = '\0';
    
    FILE *file = fopen(f_name, "rb");
    if (!file) {
        printf("Error opening file\n");
        return NULL;
    }

    size_t num_elements = fread(dst->T, sizeof(float), dst->dim[0]*dst->stride[0], file);
    if (num_elements != dst->dim[0]*dst->stride[0]) {
        printf("Error reading file\n");
        return NULL;
    }

    fclose(file);

    return dst;
}

///////////////////////////DATALOADER///////////////////////////////////
///////////////////////////DATALOADER///////////////////////////////////

FILE* LoaderINIT(const char* file_name){
    char f_name[50] = "./data/";
    int len = strlen(f_name);
    int i;
    for(i=0; file_name[i]; i++){
        f_name[i+len] = file_name[i];
    }
    f_name[i+len] = '\0';
    
    FILE *file = fopen(f_name, "rb");
    if (!file) {
        printf("Error opening file\n");
        return NULL;
    }
    return file;
}


Tensor* LoaderNEXT(Tensor* dst, FILE*file){
    if(dst->device_type){
        printf("Tensor must be on CPU\n");
        return NULL;
    }
    size_t num_elements = fread(dst->T, sizeof(float), dst->sizeTensor, file);
    if (num_elements != dst->sizeTensor) {
        printf("Error reading file\n");
        return NULL;
    }
    return dst;
}

void LoaderCLOSE(FILE* file){
    fclose(file);
}

///////////////////////////////////////////////////////////////////
/////////////////////////CrossEntropy////////////////////////////



///O->[batch_size label_len] Y->[batchsize]
float CrossEntropyLoss(Tensor* CPU_O, Tensor* CPU_Y){
    if(!CPU_O||!CPU_Y){
        printf("no Tensor.\n");
        return -1;
    }
    if(CPU_O->device_type || CPU_Y->device_type){
        printf("Tensor should be on CPU.\n");
        return -1;
    }
    if(CPU_O->dim[0] != CPU_Y->sizeTensor){
        printf("batch does not match.\n");
        return -1;
    }
    double loss = 0;
    for(int i=0; i < CPU_Y->sizeTensor;i++){
        loss -= log(CPU_O->T[CPU_O->stride[0]*i + (int)CPU_Y->T[i]]);
        
        
    }
    return loss/CPU_Y->sizeTensor;
}

__global__ void CESoftmax_deriv_(float* deriv, float* O, float* label, int O_stride, int batch_size){
    int inx = blockDim.x * blockIdx.x + threadIdx.x;//each batch
    if(inx < batch_size){
        for(int i=0; i < O_stride;i++){
            if(i == label[inx])
                deriv[O_stride * inx + i] = O[O_stride * inx + i] - 1;
            else
                deriv[O_stride * inx + i] = O[O_stride * inx + i];
        }
    }
}
Tensor* CESoftmax_deriv(Tensor* d_der_O, Tensor*d_O, Tensor* d_Y){
    if(!d_der_O || !d_O || !d_Y){
        printf("CES: no Tensor.\n");
        return NULL;
    }
    if(d_O->dim[0] != d_Y->sizeTensor){
        printf("batch does not match.\n");
        return NULL;
    }
    if(d_der_O->num_dim != d_O->num_dim){
        printf("dimention does not match.\n");
        return NULL;
    }
    hipSetDevice(d_der_O->device_type-1);
    CESoftmax_deriv_<<<(d_der_O->dim[d_der_O->num_dim - 2] + tile_SIZE - 1)/tile_SIZE, tile_SIZE>>>(d_der_O->T,d_O->T, d_Y->T, d_der_O->dim[d_der_O->num_dim - 1], d_der_O->dim[d_der_O->num_dim - 2]);
    return d_der_O;
}


int main(){
    int batch_size = 16;
    float learning_rate = 0.00002;
    int layer_dim[] = {784, 50, 30, 40, 10};
    int in_dim[] = {batch_size, layer_dim[0]};

    //==========input allocation========================
    Tensor* input = mallocTensor(in_dim, 2, 0);
    Tensor* d_input = makeTensorbyShape(input, 1);
    in_dim[0] = 784;
    in_dim[1] = batch_size;
    Tensor* d_input_t = mallocTensor(in_dim, 2, 1);

    Tensor* label = mallocTensor(&batch_size, 1, 0);
    Tensor* d_label = makeTensorbyShape(label, 1);
    //==================================================

    Tensor* W[4];
    Tensor* d_W[4];
    Tensor* b[4];
    Tensor* d_b[4];

    Tensor* d_der_W[4];
    Tensor* d_W_t[4];
    Tensor* d_der_b[4];

    Tensor* d_A[4];
    Tensor* d_der_A[4];
    Tensor* d_A_t[4];
    //이건 0-9사이에 가중치만 있으므로 가능
    //==========================weight initialization===============================
    char file_name[] = "0_init_blocks.bin";
    for(int i=0; i < sizeof(W)/sizeof(Tensor*); i++){

        file_name[0] = 2*i + '0';
        W[i] = copyTensorfromFILE(mallocTensor(layer_dim+i, 2, 0), file_name);
        d_W[i] = copyTensor(makeTensorbyShape(W[i], 1), W[i]);
        d_der_W[i] = makeTensorbyShape(W[i], 1);

        in_dim[0] = layer_dim[i+1];
        in_dim[1] = layer_dim[i];
        d_W_t[i] = mallocTensor(in_dim, 2, 1);

        file_name[0] = 2*i+1 + '0';
        b[i] = copyTensorfromFILE(mallocTensor(layer_dim+i+1, 1, 0), file_name);
        d_b[i] = copyTensor(makeTensorbyShape(b[i],1),b[i]);
        d_der_b[i] = makeTensorbyShape(b[i], 1);
        
        in_dim[0] = batch_size;
        in_dim[1] = layer_dim[i+1];
        d_A[i] = mallocTensor(in_dim, 2, 1);
        d_der_A[i] = makeTensorbyShape(d_A[i], 1);

        in_dim[0] = layer_dim[i+1];
        in_dim[1] = batch_size;
        d_A_t[i] = mallocTensor(in_dim, 2, 1);
        // infoTensor(d_A_t[i]);
    }
    
    in_dim[0] = batch_size;
    in_dim[1] = layer_dim[sizeof(layer_dim)/sizeof(int)-1];
    Tensor* O = mallocTensor(in_dim, 2, 0);
    Tensor* d_O = makeTensorbyShape(O, 1);
    //===============================================================================


    //=============dataLoader========================
    FILE * data_file, *label_file;
    //===============================================


    //==============================TRAIN===========================================
    for(int iter=0; iter < 30; iter++){//iteration

        data_file = LoaderINIT("data_norm.bin");
        label_file = LoaderINIT("label.bin");

        double loss = 0;
        int accuracy = 0;
        for(int batch=0; batch < 60000/batch_size;batch++){//batch
            d_input = copyTensor(d_input, LoaderNEXT(input, data_file));
            d_label = copyTensor(d_label, LoaderNEXT(label, label_file));//both label and d_label is written here.

            //forward pass
            d_A[0] = matmul_bias(d_A[0], d_input, d_W[0], d_b[0], 0);

            for(int i=1; i < sizeof(d_W)/sizeof(Tensor*);i++){
                ReLU_inline(d_A[i-1]);
                d_A[i] = matmul_bias(d_A[i], d_A[i-1], d_W[i], d_b[i], 0);
            }

            //loss
            d_O = softMax(d_O, d_A[sizeof(d_A)/sizeof(Tensor*)-1]);
            loss += CrossEntropyLoss(copyTensor(O, d_O), label);//cross entropy loss

            //backward pass
            if(batch < 50000/batch_size){
                d_der_A[sizeof(d_der_A)/sizeof(Tensor*) - 1] = CESoftmax_deriv(d_der_A[sizeof(d_der_A)/sizeof(Tensor*) - 1], d_O, d_label);

                for(int i = sizeof(d_der_W)/sizeof(Tensor*) - 1; i >=1; i--){   //3, 2, 1,
                    d_der_W[i] = matmul(d_der_W[i], copyTransposeTensor(d_A_t[i-1],d_A[i-1]), d_der_A[i]);
                    d_der_b[i] = rowcolwise_sum(d_der_b[i], d_der_A[i], 0);
                    
                    d_der_A[i-1] = matmul(d_der_A[i-1], d_der_A[i], copyTransposeTensor(d_W_t[i], d_W[i]));
                    d_der_A[i-1] = elementWise_Tensor(d_der_A[i-1] ,d_der_A[i-1],'m', d_A[i-1]);//dReLU가 들어가야함. d_der_A[i] = d_A[i]==0 ? 0 : d_der_A[i];
                }
                
                //update
                d_der_W[0] = matmul(d_der_W[0], copyTransposeTensor(d_input_t,d_input), d_der_A[0]);
                d_der_b[0] = rowcolwise_sum(d_der_b[0], d_A[0], 0);

                for(int i=0; i < sizeof(d_der_W)/sizeof(Tensor*); i++){
                    d_W[i] = elementWise_Tensor(d_W[i],d_W[i],'-',scalar_Tensor(d_der_W[i], '*',learning_rate));
                    d_b[i] = elementWise_Tensor(d_b[i],d_b[i],'-',scalar_Tensor(d_der_b[i], '*',learning_rate));
                }
            }else{//iter
                accuracy += accuracy_CPU(O, label);
            }
            
            print_progress(batch, 60000/batch_size);

        }
        
        printf("\nvalid acc: %d/10000 | %.2f%%\nloss: %f\n",accuracy, (float)accuracy/100,loss/(60000/batch_size));

        LoaderCLOSE(data_file);
        LoaderCLOSE(label_file);
        printf("\n");
    }

    //free Weights///////////////////////////////////////////////////////////////////////////////////////
    for(int i=0; i <sizeof(W)/sizeof(Tensor*);i++){
        freeTensor(W[i]);
        freeTensor(d_W[i]);
        freeTensor(d_der_W[i]);
        freeTensor(d_W_t[i]);

        freeTensor(b[i]);
        freeTensor(d_b[i]);
        freeTensor(d_der_b[i]);

        freeTensor(d_A[i]);
        freeTensor(d_der_A[i]);
        freeTensor(d_A_t[i]);
    }
    
    

    freeTensor(d_input);
    freeTensor(input);
    

    
    
    
}